#include "hip/hip_runtime.h"
#include "lab2.h"
#include <iostream>
#include <cassert>

using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

static const unsigned W = 854;
static const unsigned H = 480;
static const unsigned NFRAME = 24*60;
static const int permutation[] = { 151,160,137,91,90,15,
  131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
  190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
  88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
  77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
  102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
  135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
  5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
  223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9,
  129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
  251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
  49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254,
  138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
};

struct Lab2VideoGenerator::Impl {
  int t = 0;
  int blockWidth;
  dim3 threadDimension;
  dim3 blockDimension;
  double * R, * G, * B;
  int * perm;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
  impl->t = 0;
  int blockWidth = 16;
  impl->blockWidth = blockWidth;
  impl->threadDimension = dim3(blockWidth, blockWidth);
  impl->blockDimension =
    dim3(W / blockWidth + (W % blockWidth ? 1 : 0),
         H / blockWidth + (H % blockWidth ? 1 : 0));
  hipMalloc(&impl->R, sizeof(double) * H * W);
  hipMalloc(&impl->G, sizeof(double) * H * W);
  hipMalloc(&impl->B, sizeof(double) * H * W);
  hipMalloc(&impl->perm, sizeof(int) * 256);
  hipMemcpy(impl->perm, permutation, sizeof(int) * 256, hipMemcpyHostToDevice);
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

__device__ __host__ double fade(double t) {
  // 6t^5 - 15t^4 + 10t^3
  return t * t * t * (t * (t * 6.0 - 15.0) + 10.0);
}

__device__ __host__ double linearInterpolate(double l, double r, double s) {
  // s should be in [0, 1]
  return r * s + l * (1.0 - s);
}

__device__ __host__ double dotGradiant(int * p, int a, int b, int c, double dx, double dy, double dz) {
  int hashed = p[(p[(p[a & 255] + b) & 255] + c) & 255];
  switch (hashed % 12) {
    case 0x0: return dx + dy;
    case 0x1: return dx - dy;
    case 0x2: return dy - dx;
    case 0x3: return -dx - dy;
    case 0x4: return dx + dz;
    case 0x5: return dx - dz;
    case 0x6: return dz - dx;
    case 0x7: return -dx - dz;
    case 0x8: return dy + dz;
    case 0x9: return dy - dz;
    case 0xA: return dz - dy;
    case 0xB: return -dy - dz;
    // Should never reach here, but just in case
    default:  return dx + dy;
  }
}

__device__ __host__ double perlin(double x, double y, double z, int * p) {
  /* find boundary */
  int x0 = x;
  int y0 = y;
  int z0 = z;
  double xf = x - x0;
  double yf = y - y0;
  double zf = z - z0;
  double u = fade(xf);
  double v = fade(yf);
  double w = fade(zf);
  /* temparies to store intermediate result */
  double x1, x2, y1, y2;
  x1 = linearInterpolate(
        dotGradiant(p, x0, y0, z0, xf, yf, zf),
        dotGradiant(p, x0+1, y0, z0, xf-1, yf, zf),
        u
      );
  x2 = linearInterpolate(
        dotGradiant(p, x0, y0+1, z0, xf, yf-1, zf),
        dotGradiant(p, x0+1, y0+1, z0, xf-1, yf-1, zf),
        u
      );
  y1 = linearInterpolate(x1, x2, v);
  x1 = linearInterpolate(
        dotGradiant(p, x0, y0, z0+1, xf, yf, zf-1),
        dotGradiant(p, x0+1, y0, z0+1, xf-1, yf, zf-1),
        u
      );
  x2 = linearInterpolate(
        dotGradiant(p, x0, y0+1, z0+1, xf, yf-1, zf-1),
        dotGradiant(p, x0+1, y0+1, z0+1, xf-1, yf-1, zf-1),
        u
      );
  y2 = linearInterpolate(x1, x2, v);
  double z1 = linearInterpolate(y1, y2, w);
  // range is -1 ~ 1
  return z1;
}

__host__ __device__ double spin(double x, double y, double cx, double cy) {
  x = x - cx;
  y = y - cy;
  x = fabs(x);
  y = fabs(y);
  // special mossaic grid effect
  double d = x*x + y*y;
  if (sqrt(d) > 200) {
    if (y > x) return x / y;
    return y / x;
  } else {
    return d;
  }
}

__global__ void octavePerlin(double * R, double * G, double * B, int H, int W, int _t, int * perm) {
  int _x = blockIdx.x * blockDim.x + threadIdx.x;
  int _y = blockIdx.y * blockDim.y + threadIdx.y;
  if (_x >= W || _y >= H) return;
  double x, y, z;
  x = _x;
  y = _y;
  z = _t;
  z /= 24.0;
  // Octave perlin
  double ampl = 1.0;
  double freq = 0.02;
  double z1 = 0.0, w = 0.0;
  for (int i = 0; i < 7; ++i) {
    // Scale the coordinate
    z1 += ampl * fabs(perlin(x * freq, y * freq, z, perm));
    w += ampl;
    ampl *= 0.5;
    freq *= 2.0;
  }
  z1 /= w;
  int idx = _x + _y * W;
  R[idx] = sinf(z + z1 + x / 64);
  G[idx] = cosf(z1 + spin(x, y, W/2, H/2));
  B[idx] = sinf(z * 1.5 + z1 - x / 128);
}

__global__ void draw(double * R, double * G, double * B, int H, int W, uint8_t *yuv) {
  int _x = blockIdx.x * blockDim.x + threadIdx.x;
  int _y = blockIdx.y * blockDim.y + threadIdx.y;
  if (_x >= W || _y >= H) return;
  int idx = _x + _y * W;
  double r = R[idx], g = G[idx], b = B[idx];
  double Y, U, V;
  Y = +0.299 * r + 0.587 * g + 0.114 * b;
  U = -0.169 * r - 0.331 * g + 0.500 * b + 128;
  V = +0.500 * r - 0.419 * g - 0.081 * b + 128;
  yuv[idx] = 255 * Y;
  G[idx] = U;
  B[idx] = V;
  __syncthreads();
  if ((_x & 1) || (_y & 1)) {
    return;
  }
  // downsampling
  U = (G[idx] + G[idx+1] + G[idx+W] + G[idx+W+1]) / 4.0;
  V = (B[idx] + B[idx+1] + B[idx+W] + B[idx+W+1]) / 4.0;
  int _idx = _x / 2 + _y * W / 4;
  yuv[H * W + _idx] = 255 * U;
  yuv[H * W + (H * W / 4) + _idx] = 255 * V;
}

/*

  [ +0.299 +0.587 +0.114 +0   ]   [R]   [Y]
  [ -0.169 -0.331 +0.500 +128 ] x [G] = [U]
  [ +0.500 -0.419 -0.081 +128 ]   [B]   [V]
                                  [1]

  Y: [0, W*H)
  U: [W*H, W*H+W*H/4)
  V: [W*H+W*H/4, W*H+W*H/2)
 */

void Lab2VideoGenerator::Generate(uint8_t *yuv) {
  hipMemset(yuv, 128, W*H + W*H/2);
  const auto& blockDimension = impl->blockDimension;
  const auto& threadDimension = impl->threadDimension;
  octavePerlin<<<blockDimension, threadDimension>>>(impl->R, impl->G, impl->B, H, W, impl->t, impl->perm);
  CHECK;
  draw<<<blockDimension, threadDimension>>>(impl->R, impl->G, impl->B, H, W, yuv);
  CHECK;
  ++(impl->t);
  return;
}

