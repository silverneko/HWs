#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < fsize and input_gpu[idx] != '\n') {
		input_gpu[idx] = '!';
	}
}

__global__ void myTransform(char *, int);

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
        // SomeTransform<<<2, 32>>>(input_gpu, fsize);

        // set `gridDim` and `blockDim`
        int blockCount = 2;
        int threadCount = 512;
        // launch kernel
        // myTransform() changes lower case letters to upper case
        // and shift letters by the distance of 10
        // for example: 'a' -> 'K', 'b' -> 'L', 'z' -> 'J'
        myTransform<<<blockCount, threadCount>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	return 0;
}

__global__ void myTransform(char * input, int len) {
  // calculate the ID of this thread and use it as the index of the input string
  int globalID = blockIdx.x * blockDim.x + threadIdx.x;
  // grid-stride loop
  for(; globalID < len; globalID += blockDim.x * gridDim.x){
    if('a' <= input[globalID] && input[globalID] <= 'z'){
      input[globalID] += 'A' - 'a';
    }
    if('A' <= input[globalID] && input[globalID] <= 'Z'){
      input[globalID] = (((int)input[globalID] - 'A' + 10) % 26) + 'A';
    }
  }
}

