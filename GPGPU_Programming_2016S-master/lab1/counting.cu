#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void part1_init(const char *, int *, int);
__global__ void part1_kernel(const char *, int *, int, int);

void CountPosition(const char *text, int *pos, int text_size)
{
  const int maxK = 500;
  const int threadCount = 512;
  int blockCount = text_size / threadCount + 1;
  part1_init<<<blockCount, threadCount>>>(text, pos, text_size);
  hipDeviceSynchronize();
  for (int i = 2; i <= maxK; ++i) {
    blockCount = (text_size / i + 1) / threadCount + 1;
    part1_kernel<<<blockCount, threadCount>>>(text, pos, text_size, i);
    hipDeviceSynchronize();
  }
}

__global__ void part1_kernel(const char * text, int * pos, int n, int k) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  id = id * k;
  /* should be careful here as id may int overflow */
  if (id >= n) {
    return;
  }
  if (pos[id] == 0) {
    return;
  }
  if (id > 0 && pos[id-1] == k-1) {
    pos[id] = k;
    return;
  }
  int i = pos[id];
  int j = id + (k - i - 1);
  if (j+1 < n && pos[j] == k-1 && text[j+1] != '\n') {
    pos[j+1] = k;
  }
}

__global__ void part1_init(const char * text, int * pos, int n) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= n) {
    return;
  }
  pos[id] = 0;
  if (text[id] != '\n') {
    pos[id] = -1;
    if (id == 0 || text[id-1] == '\n'){
      pos[id] = 1;
    }
  }
}

template<int I>
class equals {
public:
  __device__ bool operator () (int x) { return x == I;}
};

int ExtractHead(const int *pos, int *head, int text_size)
{
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);

        auto head_end_d =
          thrust::copy_if(
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(text_size),
            pos_d,
            head_d,
            equals<1>()
          );
        nhead = head_end_d - head_d;
	hipFree(buffer);
	return nhead;
}

__global__ void part3_find_rpos(char *text, int n, int *pos, int *rpos);
__global__ void part3_string_flip(char *text, int n, int *pos, int *rpos);

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
  int n = text_size;
  int *rpos;
  hipMalloc(&rpos, n*sizeof(int));
  hipMemset(rpos, 0, n*sizeof(int));
  int threadCount = 256;
  int blockCount = 1 + n / threadCount;
  part3_find_rpos<<<blockCount, threadCount>>>(text, n, pos, rpos);
  hipDeviceSynchronize();
  part3_string_flip<<<blockCount, threadCount>>>(text, n, pos, rpos);
  hipDeviceSynchronize();
}

__global__ void part3_find_rpos(char *text, int n, int *pos, int *rpos) {
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  if (ID >= n || pos[ID] == 0) {
    return;
  }
  int lb = ID, rb = ID+501;
  while (lb != rb-1) {
    int mid = (lb+rb) / 2;
    if (mid >= n) {
      rb = mid;
    } else if (mid - ID != pos[mid] - pos[ID]) {
      rb = mid;
    } else {
      lb = mid;
    }
  }
  rpos[ID] = lb - pos[ID] + 1;
}

__global__ void part3_string_flip(char *text, int n, int *pos, int *rpos) {
  int ID = blockIdx.x * blockDim.x + threadIdx.x;
  if (ID >= n || ID >= rpos[ID]) {
    return;
  }
  int pos1 = ID, pos2 = rpos[ID];
  char c = text[pos1];
  text[pos1] = text[pos2];
  text[pos2] = c;
}

