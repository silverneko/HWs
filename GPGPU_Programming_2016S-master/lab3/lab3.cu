#include "hip/hip_runtime.h"
#include <cstdio>
#include <functional>

#include "lab3.h"

#ifdef DEBUG
#include "Timer.h"
#endif

using namespace std;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
  const float *background,
  const float *target,
  const float *mask,
  float *output,
  const int wb, const int hb, const int wt, const int ht,
  const int oy, const int ox
  )
{
  const int yt = blockIdx.y * blockDim.y + threadIdx.y;
  const int xt = blockIdx.x * blockDim.x + threadIdx.x;
  const int curt = wt*yt+xt;
  if (yt < ht and xt < wt and mask[curt] > 127.0f) {
    const int yb = oy+yt, xb = ox+xt;
    const int curb = wb*yb+xb;
    if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
      output[curb*3+0] = target[curt*3+0];
      output[curb*3+1] = target[curt*3+1];
      output[curb*3+2] = target[curt*3+2];
    }
  }
}

class RGB {
public:
  float * R, * G, * B;
};

__global__ void calculateFixed(
  RGB background,
  RGB target,
  const float * mask,
  RGB output,
  const int wb, const int hb, const int wt, const int ht,
  const int oy, const int ox
  )
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int idy = blockDim.y * blockIdx.y + threadIdx.y;
  const int curt = wt * idy + idx;
  if (idx >= wt || idy >= ht || mask[curt] < 127.0f) {
    return;
  }
  float result[3];
  result[0] = 4*target.R[curt];
  result[1] = 4*target.G[curt];
  result[2] = 4*target.B[curt];
  const int dir[4][2] = {{1, 1}, {1, -1}, {-1, 1}, {-1, -1}};
  for (int i = 0; i < 4; ++i) {
    const int nx = idx + dir[i][0];
    const int ny = idy + dir[i][1];
    const int curn = wt*ny + nx;
    if (nx >= 0 && ny >= 0 && nx < wt && ny < ht) {
      result[0] -= target.R[curn];
      result[1] -= target.G[curn];
      result[2] -= target.B[curn];
    } else {
      result[0] -= target.R[curt];
      result[1] -= target.G[curt];
      result[2] -= target.B[curt];
    }
    if ((nx < 0 || ny < 0 || nx >= wt || ny >= ht) || mask[curn] < 127.0f) {
      const int bx = nx + ox;
      const int by = ny + oy;
      const int curb = wb * by + bx;
      result[0] += background.R[curb];
      result[1] += background.G[curb];
      result[2] += background.B[curb];
    }
  }
  output.R[curt] = result[0];
  output.G[curt] = result[1];
  output.B[curt] = result[2];
}

__global__ void calculateJacobi(
  RGB fixed,
  const float * mask,
  RGB target,
  RGB output,
  const int wt,
  const int ht
  )
{
  const int idx = blockDim.x * blockIdx.x + threadIdx.x;
  const int idy = blockDim.y * blockIdx.y + threadIdx.y;
  const int curt = wt * idy + idx;
  if (idx >= wt || idy >= ht || mask[curt] < 127.0f) {
    return;
  }
  float result[3];
  result[0] = fixed.R[curt];
  result[1] = fixed.G[curt];
  result[2] = fixed.B[curt];
  const int dir[4][2] = {{1, 1}, {1, -1}, {-1, 1}, {-1, -1}};
  for (int i = 0; i < 4; ++i) {
    const int nx = idx + dir[i][0];
    const int ny = idy + dir[i][1];
    const int curn = wt*ny + nx;
    if (nx >= 0 && nx < wt && ny >= 0 && ny < ht && mask[curn] > 127.0f) {
      result[0] += target.R[curn];
      result[1] += target.G[curn];
      result[2] += target.B[curn];
    }
  }
  output.R[curt] = result[0] / 4;
  output.G[curt] = result[1] / 4;
  output.B[curt] = result[2] / 4;
}

__global__ void transpose2RGBinternal(const float * ch, int nsize, RGB rgb) {
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= nsize) {
    return;
  }
  rgb.R[id] = ch[3*id + 0];
  rgb.G[id] = ch[3*id + 1];
  rgb.B[id] = ch[3*id + 2];
}

__global__ void transposeFromRGBinternal(RGB rgb, int nsize, float * ch) {
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= nsize) {
    return;
  }
  ch[3*id + 0] = rgb.R[id];
  ch[3*id + 1] = rgb.G[id];
  ch[3*id + 2] = rgb.B[id];
}

void transpose2RGB(const float * ch, int nsize, RGB rgb) {
  transpose2RGBinternal<<<CeilDiv(nsize, 512), 512>>>(ch, nsize, rgb);
}

void transposeFromRGB(RGB rgb, int nsize, float * ch) {
  transposeFromRGBinternal<<<CeilDiv(nsize, 512), 512>>>(rgb, nsize, ch);
}

void newRGB(RGB * rgb, int nsize) {
  hipMalloc(&rgb->R, nsize * sizeof(float));
  hipMalloc(&rgb->G, nsize * sizeof(float));
  hipMalloc(&rgb->B, nsize * sizeof(float));
}

void freeRGB(RGB rgb) {
  hipFree(rgb.R);
  hipFree(rgb.G);
  hipFree(rgb.B);
}

void PoissonImageCloning(
  const float *background,
  const float *target,
  const float *mask,
  float *output,
  const int wb, const int hb, const int wt, const int ht,
  const int oy, const int ox
  )
{
  const int nsize = wt * ht;
  RGB bgRGB, fgRGB;
  newRGB(&bgRGB, wb * hb);
  newRGB(&fgRGB, nsize);
  transpose2RGB(background, wb * hb, bgRGB);
  transpose2RGB(target, nsize, fgRGB);

  RGB fixed, buf1, buf2;
  newRGB(&fixed, nsize);
  newRGB(&buf1, nsize);
  newRGB(&buf2, nsize);

#ifdef DEBUG
  /* TIMER */
  hipDeviceSynchronize();
  Timer timer;
  timer.Start();
  /* TIMER */
#endif

  dim3 gridDimension(CeilDiv(wt, 32), CeilDiv(ht, 16)), blockDimension(32, 16);
  calculateFixed <<<gridDimension, blockDimension>>> (
    bgRGB, fgRGB, mask, fixed, wb, hb, wt, ht, oy, ox
  );

  hipMemcpy(buf1.R, fgRGB.R, nsize * sizeof(float), hipMemcpyDeviceToDevice);
  hipMemcpy(buf1.G, fgRGB.G, nsize * sizeof(float), hipMemcpyDeviceToDevice);
  hipMemcpy(buf1.B, fgRGB.B, nsize * sizeof(float), hipMemcpyDeviceToDevice);
  for (int i = 0; i < 20000; ++i) {
    calculateJacobi <<<gridDimension, blockDimension>>> (
      fixed, mask, buf1, buf2, wt, ht
    );
    swap(buf1, buf2);
  }

  float * buf3;
  hipMalloc(&buf3, 3 * nsize * sizeof(float));
  transposeFromRGB(buf1, nsize, buf3);
  hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
  SimpleClone <<<gridDimension, blockDimension>>>(
    background, buf3, mask, output,
    wb, hb, wt, ht, oy, ox
  );

#ifdef DEBUG
  /* TIMER */
  hipDeviceSynchronize();
  timer.Pause();
  printf_timer(timer);
  /* TIMER */
#endif

  hipFree(buf3);
  freeRGB(buf2);
  freeRGB(buf1);
  freeRGB(fixed);
  freeRGB(fgRGB);
  freeRGB(bgRGB);
}
